
#include <hip/hip_runtime.h>
/* -*- mode: c++ -*- */

extern "C" __global__ void
filter(const float * __restrict__ packed_input,
       int nInputPlanes,
       float * __restrict__ packed_output,
       int nOutputPlanes,
       const float * __restrict__ biases,
       unsigned int hsz,
       unsigned int wsz,
       const float * __restrict__ weight)
{
	extern __shared__ float shared_buf[];

	unsigned int yi = blockIdx.x;

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == wsz-1) {
		in2p = inp;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	for (int xi=0; xi<wsz; xi++) {
		/*for (unsigned int op=0; op<nOutputPlanes; op++) thread */
		{
			int op = threadIdx.x;
			float sum=0;
			for (unsigned int ip=0; ip<nInputPlanes; ip++) {
				float i00, i01, i02;
				float i10, i11, i12;
				float i20, i21, i22;

				i01 = in01[xi*nInputPlanes+ip];
				i11 = in11[xi*nInputPlanes+ip];
				i21 = in21[xi*nInputPlanes+ip];

				if (xi == 0) {
					i00 = i01;
					i10 = i11;
					i20 = i21;
				} else {
					i00 = in01[(xi-1)*nInputPlanes+ip];
					i10 = in11[(xi-1)*nInputPlanes+ip];
					i20 = in21[(xi-1)*nInputPlanes+ip];
				}

				if (xi == wsz-1) {
					i02 = i01;
					i12 = i11;
					i22 = i21;
				} else {
					i02 = in01[(xi+1)*nInputPlanes+ip];
					i12 = in11[(xi+1)*nInputPlanes+ip];
					i22 = in21[(xi+1)*nInputPlanes+ip];
				}

				sum += i00 * weight[(9*ip+0) * 128 + op];
				sum += i01 * weight[(9*ip+1) * 128 + op];
				sum += i02 * weight[(9*ip+2) * 128 + op];

				sum += i10 * weight[(9*ip+3) * 128 + op];
				sum += i11 * weight[(9*ip+4) * 128 + op];
				sum += i12 * weight[(9*ip+5) * 128 + op];

				sum += i20 * weight[(9*ip+6) * 128 + op];
				sum += i21 * weight[(9*ip+7) * 128 + op];
				sum += i22 * weight[(9*ip+8) * 128 + op];
			}

			float v = sum;
			float bv = biases[op];
			v += bv;
			float mtz = max(v,0.0f);
			float ltz = min(v,0.0f);

			v = ltz * 0.1f + mtz;

			float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;

			out[op] = v;
		}
	}
}
