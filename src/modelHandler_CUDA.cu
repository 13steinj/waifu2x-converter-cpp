
#include <hip/hip_runtime.h>
/* -*- mode: c++ -*- */

#define UNROLL9(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\
	F(8);					\


#define UNROLL8x3x3(F)				\
	F(0,0,0);				\
	F(0,0,1);				\
	F(0,0,2);				\
	F(0,1,0);				\
	F(0,1,1);				\
	F(0,1,2);				\
	F(0,2,0);				\
	F(0,2,1);				\
	F(0,2,2);				\
						\
	F(1,0,0);				\
	F(1,0,1);				\
	F(1,0,2);				\
	F(1,1,0);				\
	F(1,1,1);				\
	F(1,1,2);				\
	F(1,2,0);				\
	F(1,2,1);				\
	F(1,2,2);				\
						\
	F(2,0,0);				\
	F(2,0,1);				\
	F(2,0,2);				\
	F(2,1,0);				\
	F(2,1,1);				\
	F(2,1,2);				\
	F(2,2,0);				\
	F(2,2,1);				\
	F(2,2,2);				\
						\
	F(3,0,0);				\
	F(3,0,1);				\
	F(3,0,2);				\
	F(3,1,0);				\
	F(3,1,1);				\
	F(3,1,2);				\
	F(3,2,0);				\
	F(3,2,1);				\
	F(3,2,2);				\
						\
	F(4,0,0);				\
	F(4,0,1);				\
	F(4,0,2);				\
	F(4,1,0);				\
	F(4,1,1);				\
	F(4,1,2);				\
	F(4,2,0);				\
	F(4,2,1);				\
	F(4,2,2);				\
						\
	F(5,0,0);				\
	F(5,0,1);				\
	F(5,0,2);				\
	F(5,1,0);				\
	F(5,1,1);				\
	F(5,1,2);				\
	F(5,2,0);				\
	F(5,2,1);				\
	F(5,2,2);				\
						\
	F(6,0,0);				\
	F(6,0,1);				\
	F(6,0,2);				\
	F(6,1,0);				\
	F(6,1,1);				\
	F(6,1,2);				\
	F(6,2,0);				\
	F(6,2,1);				\
	F(6,2,2);				\
						\
	F(7,0,0);				\
	F(7,0,1);				\
	F(7,0,2);				\
	F(7,1,0);				\
	F(7,1,1);				\
	F(7,1,2);				\
	F(7,2,0);				\
	F(7,2,1);				\
	F(7,2,2);				\

#define UNROLL8(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\


#define UNROLL8x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\


#define UNROLL10x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
	F(0,8);					\
	F(0,9);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
	F(1,8);					\
	F(1,9);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\
	F(2,8);					\
	F(2,9);					\


#define BLOCK_SIZE 8

template <int nInputPlanes>
__device__ void
filter(const float * __restrict__ packed_input,
       float * __restrict__ packed_output,
       int nOutputPlanes,
       const float * __restrict__ biases,
       unsigned int hsz,
       unsigned int wsz,
       const float * __restrict__ weight)
{
	extern __shared__ float shared_buf[];

	unsigned int yi = blockIdx.x;

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == hsz-1) {
		in2p = in1p;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	float *shared_ptr = shared_buf;
	float *in_block0_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block1_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block2_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);

	float *in_block0 = in_block0_base + nInputPlanes;
	float *in_block1 = in_block1_base + nInputPlanes;
	float *in_block2 = in_block2_base + nInputPlanes;
	int lid = threadIdx.x;
	float bv = biases[lid];

	for (int xi0=0; xi0<wsz; xi0+=BLOCK_SIZE) {

		/*for (unsigned int op=0; op<nOutputPlanes; op++) thread */
		{
			int op = lid;
			int rem = wsz - xi0;
			__syncthreads();
			if (lid < nInputPlanes/2) {
				int bi;
				int lid2 = lid*2;
				for (bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0 + bi;
					if (xi == wsz) {
						break;
					}

					/* load to shared */
					*(float2*)&in_block0[bi*nInputPlanes + lid2] = *(float2*)&in01[xi*nInputPlanes + lid2];
					*(float2*)&in_block1[bi*nInputPlanes + lid2] = *(float2*)&in11[xi*nInputPlanes + lid2];
					*(float2*)&in_block2[bi*nInputPlanes + lid2] = *(float2*)&in21[xi*nInputPlanes + lid2];
				}

				{
					int xi = xi0 + bi;
					if (xi == wsz) {
						*(float2*)&in_block0[bi*(int)nInputPlanes + lid2] = *(float2*)&in01[(xi-1)*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[bi*(int)nInputPlanes + lid2] = *(float2*)&in11[(xi-1)*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[bi*(int)nInputPlanes + lid2] = *(float2*)&in21[(xi-1)*(int)nInputPlanes + lid2];
					} else {
						*(float2*)&in_block0[bi*(int)nInputPlanes + lid2] = *(float2*)&in01[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[bi*(int)nInputPlanes + lid2] = *(float2*)&in11[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[bi*(int)nInputPlanes + lid2] = *(float2*)&in21[xi*(int)nInputPlanes + lid2];
					}
				}

				{
					int xi = xi0-1;
					if (xi == -1) {
						*(float2*)&in_block0[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in01[lid2];
						*(float2*)&in_block1[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in11[lid2];
						*(float2*)&in_block2[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in21[lid2];
					} else {
						*(float2*)&in_block0[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in01[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in11[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in21[xi*(int)nInputPlanes + lid2];
					}
				}
			}
			__syncthreads();

			if (rem >= BLOCK_SIZE) {
#define DECL_PTR(y,x)		float *p##y##x = &in_block##y[nInputPlanes * (x-1)];

				UNROLL10x3(DECL_PTR);

				float sum0 = 0;
				float sum1 = 0;
				float sum2 = 0;
				float sum3 = 0;

				float sum4 = 0;
				float sum5 = 0;
				float sum6 = 0;
				float sum7 = 0;

				{
					const float *w0 = weight + lid;

					for (int ip = 0; ip < nInputPlanes; ip++) {
#define LOAD_INPUT2(y,x)			float2 i##y##x##_2 = *(float2*)&p##y##x[ip];

						UNROLL10x3(LOAD_INPUT2);

#define LOAD_COEF(X)				float w_##X = w[X * 128];

#define CALC(IDX,Y,I0,I1,I2,I3,I4,I5,I6,I7)				\
						sum0 += w_##IDX * i##Y##I0; \
						sum1 += w_##IDX * i##Y##I1; \
						sum2 += w_##IDX * i##Y##I2; \
						sum3 += w_##IDX * i##Y##I3; \
						sum4 += w_##IDX * i##Y##I4; \
						sum5 += w_##IDX * i##Y##I5; \
						sum6 += w_##IDX * i##Y##I6; \
						sum7 += w_##IDX * i##Y##I7;


						{
#define LOAD_INPUT1X(Y,X)				float i##Y##X = i##Y##X##_2.x;

							UNROLL10x3(LOAD_INPUT1X);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

						ip++;
						{
#define LOAD_INPUT1Y(Y,X)				float i##Y##X = i##Y##X##_2.y;

							UNROLL10x3(LOAD_INPUT1Y);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

					}

#define RELU(BI)							\
					{				\
						float *out = packed_output + (yi*wsz + (xi0+BI))*nOutputPlanes; \
									\
						{			\
							int opIndex = lid; \
							float v = sum##BI; \
							v += bv;	\
									\
							float mtz = max(v, 0.0f); \
							float ltz = min(v, 0.0f); \
									\
							v = ltz * 0.1f + mtz; \
									\
							out[opIndex] = v; \
						}			\
					}

					UNROLL8(RELU);
				}
			} else {
				for (int bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0+bi;
					if (xi == wsz) {
						break;
					}

					const float *w0 = weight + lid;
					float sum = 0;

					for (int ip=0; ip<nInputPlanes; ip++) {
						float i00, i01, i02;
						float i10, i11, i12;
						float i20, i21, i22;

						i00 = in_block0[(bi-1)*nInputPlanes+ip];
						i10 = in_block1[(bi-1)*nInputPlanes+ip];
						i20 = in_block2[(bi-1)*nInputPlanes+ip];

						i01 = in_block0[bi*nInputPlanes+ip];
						i11 = in_block1[bi*nInputPlanes+ip];
						i21 = in_block2[bi*nInputPlanes+ip];

						i02 = in_block0[(bi+1)*nInputPlanes+ip];
						i12 = in_block1[(bi+1)*nInputPlanes+ip];
						i22 = in_block2[(bi+1)*nInputPlanes+ip];

						const float *w = w0;
						sum += w[(9*ip+0) * 128]*i00;
						sum += w[(9*ip+1) * 128]*i01;
						sum += w[(9*ip+2) * 128]*i02;

						sum += w[(9*ip+3) * 128]*i10;
						sum += w[(9*ip+4) * 128]*i11;
						sum += w[(9*ip+5) * 128]*i12;

						sum += w[(9*ip+6) * 128]*i20;
						sum += w[(9*ip+7) * 128]*i21;
						sum += w[(9*ip+8) * 128]*i22;
					}

					float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;
					{
						float v = sum;
						v += bv;

						float mtz = max(v, 0.0f);
						float ltz = min(v, 0.0f);

						v = ltz * 0.1f + mtz;
						out[op] = v;
					}
				}
			}
		}
	}
}

extern "C" __global__ void
filter_i32(const float * __restrict__ packed_input,
	   float * __restrict__ packed_output,
	   int nOutputPlanes,
	   const float * __restrict__ biases,
	   unsigned int hsz,
	   unsigned int wsz,
	   const float * __restrict__ weight)
{
	filter<32>(packed_input, packed_output, nOutputPlanes, biases, hsz, wsz, weight);
}

extern "C" __global__ void
filter_i64(const float * __restrict__ packed_input,
	   float * __restrict__ packed_output,
	   int nOutputPlanes,
	   const float * __restrict__ biases,
	   unsigned int hsz,
	   unsigned int wsz,
	   const float * __restrict__ weight)
{
	filter<64>(packed_input, packed_output, nOutputPlanes, biases, hsz, wsz, weight);
}

extern "C" __global__ void
filter_i128(const float * __restrict__ packed_input,
	    float * __restrict__ packed_output,
	    int nOutputPlanes,
	    const float * __restrict__ biases,
	    unsigned int hsz,
	    unsigned int wsz,
	    const float * __restrict__ weight)
{
	filter<128>(packed_input, packed_output, nOutputPlanes, biases, hsz, wsz, weight);
}

static __device__ float
warp_sum(float v) {
    v += __shfl_down(v, 1);
    v += __shfl_down(v, 2);
    v += __shfl_down(v, 4);
    v += __shfl_down(v, 8);
    v += __shfl_down(v, 16);

    return v;
}

extern "C" __global__ void
filter_i128_o128(const float * __restrict__ packed_input,
		 float * __restrict__ packed_output,
		 const float * __restrict__ biases,
		 unsigned int hsz,
		 unsigned int wsz,
		 const float * __restrict__ weight)
{
	int nInputPlanes = 128;
	int nOutputPlanes = 128;

	/* 1024 thread
	 *  128 input plane x 32 output plane / block  (147KB regs)
	 *   4  input plane                   / thread (36 regs), 1output plane = 32thread
	 *
	 * 4block / pixel
	 *
	 * block  [yi       , op32(0-3) ]
	 * thread [op1(0-31), ip(0-31)  ] (1024thread)
	 *
	 * op       = op32*32 + op1
	 * ip       = (ip*4+0, ip*4+1, ip*4+2, ip*4+3)
	 */
	int yi = blockIdx.y;
	int op32 = blockIdx.x;
	int op1 = threadIdx.y;
	int ip0 = threadIdx.x*4;

	int lid = threadIdx.x + threadIdx.y*32;

	int op = op32 * 32 + op1;

	float w000 = weight[(ip0*9 + 0) * nOutputPlanes + op];
	float w001 = weight[(ip0*9 + 1) * nOutputPlanes + op];
	float w002 = weight[(ip0*9 + 2) * nOutputPlanes + op];
	float w010 = weight[(ip0*9 + 3) * nOutputPlanes + op];
	float w011 = weight[(ip0*9 + 4) * nOutputPlanes + op];
	float w012 = weight[(ip0*9 + 5) * nOutputPlanes + op];
	float w020 = weight[(ip0*9 + 6) * nOutputPlanes + op];
	float w021 = weight[(ip0*9 + 7) * nOutputPlanes + op];
	float w022 = weight[(ip0*9 + 8) * nOutputPlanes + op];

	float w100 = weight[((ip0+1)*9 + 0) * nOutputPlanes + op];
	float w101 = weight[((ip0+1)*9 + 1) * nOutputPlanes + op];
	float w102 = weight[((ip0+1)*9 + 2) * nOutputPlanes + op];
	float w110 = weight[((ip0+1)*9 + 3) * nOutputPlanes + op];
	float w111 = weight[((ip0+1)*9 + 4) * nOutputPlanes + op];
	float w112 = weight[((ip0+1)*9 + 5) * nOutputPlanes + op];
	float w120 = weight[((ip0+1)*9 + 6) * nOutputPlanes + op];
	float w121 = weight[((ip0+1)*9 + 7) * nOutputPlanes + op];
	float w122 = weight[((ip0+1)*9 + 8) * nOutputPlanes + op];

	float w200 = weight[((ip0+2)*9 + 0) * nOutputPlanes + op];
	float w201 = weight[((ip0+2)*9 + 1) * nOutputPlanes + op];
	float w202 = weight[((ip0+2)*9 + 2) * nOutputPlanes + op];
	float w210 = weight[((ip0+2)*9 + 3) * nOutputPlanes + op];
	float w211 = weight[((ip0+2)*9 + 4) * nOutputPlanes + op];
	float w212 = weight[((ip0+2)*9 + 5) * nOutputPlanes + op];
	float w220 = weight[((ip0+2)*9 + 6) * nOutputPlanes + op];
	float w221 = weight[((ip0+2)*9 + 7) * nOutputPlanes + op];
	float w222 = weight[((ip0+2)*9 + 8) * nOutputPlanes + op];

	float w300 = weight[((ip0+3)*9 + 0) * nOutputPlanes + op];
	float w301 = weight[((ip0+3)*9 + 1) * nOutputPlanes + op];
	float w302 = weight[((ip0+3)*9 + 2) * nOutputPlanes + op];
	float w310 = weight[((ip0+3)*9 + 3) * nOutputPlanes + op];
	float w311 = weight[((ip0+3)*9 + 4) * nOutputPlanes + op];
	float w312 = weight[((ip0+3)*9 + 5) * nOutputPlanes + op];
	float w320 = weight[((ip0+3)*9 + 6) * nOutputPlanes + op];
	float w321 = weight[((ip0+3)*9 + 7) * nOutputPlanes + op];
	float w322 = weight[((ip0+3)*9 + 8) * nOutputPlanes + op];

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == hsz-1) {
		in2p = in1p;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	__shared__ float in00_block_buf[128];
	__shared__ float in01_block_buf[128];
	__shared__ float in02_block_buf[128];

	__shared__ float in10_block_buf[128];
	__shared__ float in11_block_buf[128];
	__shared__ float in12_block_buf[128];

	__shared__ float in20_block_buf[128];
	__shared__ float in21_block_buf[128];
	__shared__ float in22_block_buf[128];

	float *in00_block = in00_block_buf;
	float *in01_block = in01_block_buf;
	float *in02_block = in02_block_buf;

	float *in10_block = in10_block_buf;
	float *in11_block = in11_block_buf;
	float *in12_block = in12_block_buf;

	float *in20_block = in20_block_buf;
	float *in21_block = in21_block_buf;
	float *in22_block = in22_block_buf;

	float bv = biases[op];

	if (lid < 128) {
		float v0 = in0p[lid];
		float v1 = in1p[lid];
		float v2 = in2p[lid];

		in01_block[lid] = in02_block_buf[lid] = v0;
		in11_block[lid] = in12_block_buf[lid] = v1;
		in21_block[lid] = in22_block_buf[lid] = v2;
	}

	for (int xi=0; xi<wsz; xi++) {
		float *tmp0 = in00_block;
		float *tmp1 = in10_block;
		float *tmp2 = in20_block;

		in00_block = in01_block; in01_block = in02_block; in02_block = tmp0;
		in10_block = in11_block; in11_block = in12_block; in12_block = tmp1;
		in20_block = in21_block; in21_block = in22_block; in22_block = tmp2;

		if (xi == wsz-1) {
			in02_block = in01_block;
			in12_block = in11_block;
			in22_block = in21_block;
		} else {
			__syncthreads();
			if (lid < nInputPlanes) {
				tmp0[lid] = in0p[(xi+1)*nInputPlanes + lid];
				tmp1[lid] = in1p[(xi+1)*nInputPlanes + lid];
				tmp2[lid] = in2p[(xi+1)*nInputPlanes + lid];
			}
			__syncthreads();
		}
#if 1
		float sum = 0;
#define CONVOLVE(I) {							\
			float v00, v01, v02;				\
			float v10, v11, v12;				\
			float v20, v21, v22;				\
									\
			v00 = in00_block[ip0 + I];			\
			v10 = in10_block[ip0 + I];			\
			v20 = in20_block[ip0 + I];			\
									\
			v01 = in01_block[ip0 + I];			\
			v11 = in11_block[ip0 + I];			\
			v21 = in21_block[ip0 + I];			\
									\
			v02 = in02_block[ip0 + I];			\
			v12 = in12_block[ip0 + I];			\
			v22 = in22_block[ip0 + I];			\
									\
			sum += w##I##00 * v00;				\
			sum += w##I##01 * v01;				\
			sum += w##I##02 * v02;				\
									\
			sum += w##I##10 * v10;				\
			sum += w##I##11 * v11;				\
			sum += w##I##12 * v12;				\
									\
			sum += w##I##20 * v20;				\
			sum += w##I##21 * v21;				\
			sum += w##I##22 * v22;				\
		}

		CONVOLVE(0);
		CONVOLVE(1);
		CONVOLVE(2);
		CONVOLVE(3);

		sum = warp_sum(sum);

		if (ip0 == 0) {
			float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;
			float v = sum;
			v += bv;

			float mtz = max(v, 0.0f);
			float ltz = min(v, 0.0f);

			v = ltz * 0.1f + mtz;
			out[op] = v;
		}

#else
		float sum = 0;
		for (int ip=0; ip<nInputPlanes; ip++) {
			float v00, v01, v02;
			float v10, v11, v12;
			float v20, v21, v22;

			//v01 = in0p[xi*nInputPlanes + ip];
			v01 = in01_block[ip];
			v11 = in1p[xi*nInputPlanes + ip];
			v21 = in2p[xi*nInputPlanes + ip];

			if (xi == 0) {
				v00 = v01;
				v10 = v11;
				v20 = v21;
			} else {
				v00 = in0p[(xi-1)*nInputPlanes + ip];
				v10 = in1p[(xi-1)*nInputPlanes + ip];
				v20 = in2p[(xi-1)*nInputPlanes + ip];
			}

			if (xi == wsz-1) {
				v02 = v01;
				v12 = v11;
				v22 = v21;
			} else {
				v02 = in0p[(xi+1)*nInputPlanes + ip];
				v12 = in1p[(xi+1)*nInputPlanes + ip];
				v22 = in2p[(xi+1)*nInputPlanes + ip];
			}

			sum += weight[ip*128*9 + op + 0*128] * v00;
			sum += weight[ip*128*9 + op + 1*128] * v01;
			sum += weight[ip*128*9 + op + 2*128] * v02;

			sum += weight[ip*128*9 + op + 3*128] * v10;
			sum += weight[ip*128*9 + op + 4*128] * v11;
			sum += weight[ip*128*9 + op + 5*128] * v12;

			sum += weight[ip*128*9 + op + 6*128] * v20;
			sum += weight[ip*128*9 + op + 7*128] * v21;
			sum += weight[ip*128*9 + op + 8*128] * v22;
		}

		if (ip0 == 0) {
			float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;
			float v = sum;
			v += bv;

			float mtz = max(v, 0.0f);
			float ltz = min(v, 0.0f);

			v = ltz * 0.1f + mtz;
			out[op] = v;
		}
#endif

	}
}
